#include "hip/hip_runtime.h"
#include "blur.cuh"
#include "ErrorCheck.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>

__device__
void cuda_blur_kernel_convolution(uint thread_index, const float* gpu_raw_data,
                                  const float* gpu_blur_v, float* gpu_out_data,
                                  const unsigned int n_frames,
                                  const unsigned int blur_v_size) {
    // TODO: Implement the necessary convolution function that should be
    //       completed for each thread_index. Use the CPU implementation in
    //       blur.cpp as a reference.
}

__global__
void cuda_blur_kernel(const float *gpu_raw_data, const float *gpu_blur_v,
                      float *gpu_out_data, int n_frames, int blur_v_size) {
    // TODO: Compute the current thread index.
    uint thread_index;

    // TODO: Update the while loop to handle all indices for this thread.
    //       Remember to advance the index as necessary.
    while (false) {
        // Do computation for this thread index
        cuda_blur_kernel_convolution(thread_index, gpu_raw_data,
                                     gpu_blur_v, gpu_out_data,
                                     n_frames, blur_v_size);
        // TODO: Update the thread index
    }
}


float cuda_call_blur_kernel(const unsigned int blocks,
                            const unsigned int threads_per_block,
                            const float *raw_data,
                            const float *blur_v,
                            float *out_data,
                            const unsigned int n_frames,
                            const unsigned int blur_v_size) {
    // Use the CUDA machinery for recording time
    hipEvent_t start_gpu, stop_gpu;
    float time_milli = -1;
    checkCuda(hipEventCreate(&start_gpu));
    checkCuda(hipEventCreate(&stop_gpu));
    checkCuda(hipEventRecord(start_gpu));

    // TODO: Allocate GPU memory for the raw input data (either audio file
    //       data or randomly generated data. The data is of type float and
    //       has n_frames elements. Then copy the data in raw_data into the
    //       GPU memory you allocated.
    float* gpu_raw_data;
    // ALWAYS use checkCuda (or your own error checking function) for every cuda call

    // TODO: Allocate GPU memory for the impulse signal (for now global GPU
    //       memory is fine. The data is of type float and has blur_v_size
    //       elements. Then copy the data in blur_v into the GPU memory you
    //       allocated.
    float* gpu_blur_v;

    // TODO: Allocate GPU memory to store the output audio signal after the
    //       convolution. The data is of type float and has n_frames elements.
    //       Initialize the data as necessary.
    float* gpu_out_data;
    
    // TODO: Appropriately call the kernel function.

    // Check for errors on kernel call
    // Always include an error check after every kernel call
    checkCuda(hipGetLastError());

    // TODO: Now that kernel calls have finished, copy the output signal
    //       back from the GPU to host memory. (We store this channel's result
    //       in out_data on the host.)

    // TODO: Now that we have finished our computations on the GPU, free the
    //       GPU resources.

    // Stop the recording timer and return the computation time
    checkCuda(hipEventRecord(stop_gpu));
    checkCuda(hipEventSynchronize(stop_gpu));
    checkCuda(hipEventElapsedTime(&time_milli, start_gpu, stop_gpu));
    return time_milli;
}
